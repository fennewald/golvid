#include "hip/hip_runtime.h"
#include "src/sim.cuh"

#include "src/vec.cuh"

#include <hiprand/hiprand_kernel.h>

#include <cstdint>

inline __device__ uint32_t idx_1(void) {
	return (blockIdx.x * blockDim.x) + threadIdx.x;
}

inline __device__ IVec2 idx_2(void) {
	return IVec2{
	    static_cast<int>((blockIdx.x * blockDim.x) + threadIdx.x),
	    static_cast<int>((blockIdx.y * blockDim.y) + threadIdx.y),
	};
}

template<typename T>
inline __device__ T * pitch_ptr(T * base, IVec2 idx, int pitch) {
	uint32_t x = idx.x % params::width;
	uint32_t y = idx.y % params::height;
	return ((T *)((char *)base + (y * pitch))) + x;
}

namespace initialize {

__global__ void k_cells(Cell * cells, int pitch) {
	auto coords = idx_2();
	if (coords.x >= static_cast<int>(params::width)) return;
	if (coords.y >= static_cast<int>(params::height)) return;

	Cell * output = pitch_ptr(cells, coords, pitch);
	output->x = 0;
}


__global__ void k_agents(float * x, float * y, float * dir) {
	auto idx = idx_1();
	if (idx >= params::n_agents) return;

	hiprandState rng;
	hiprand_init(10881, idx, 0, &rng);

	x[idx] = (float)(hiprand(&rng) % params::width);
	y[idx] = (float)(hiprand(&rng) % params::height);
	dir[idx] = 45 / (2 * M_PI);
}

__host__ void cells(Cell * cells, int pitch) {
	k_cells<<<params::cells_grid_dim, params::cells_block_dim>>>(cells, pitch);
}

__host__ void agents(float * x, float * y, float * dir) {
	k_agents<<<params::agent_grid_dim, params::agent_block_dim>>>(x, y, dir);
}

}  // namespace initialize


inline __device__ void render_cell(const Cell * cell, Pixel * pixel) {
	auto v = cell->x;
	if (v > 0xff) {
		pixel->r = 0xff;
		pixel->g = 0xff;
		pixel->b = 0xff;
	} else {
		pixel->r = cell->x % 0xff;
		pixel->g = 0;
		pixel->b = 0;
	}
	pixel->a = 0xff;
}

__global__ void
k_render(const Cell * cells, int cell_pitch, Pixel * pixels, int pix_pitch) {
	auto coords = idx_2();
	if (coords.x >= static_cast<int>(params::width)) return;
	if (coords.y >= static_cast<int>(params::height)) return;

	render_cell(
	    pitch_ptr(cells, coords, cell_pitch),
	    pitch_ptr(pixels, coords, pix_pitch));
}

__host__ void
render(const Cell * cells, int cell_pitch, Pixel * pixels, int pix_pitch) {
	k_render<<<params::cells_grid_dim, params::cells_block_dim>>>(
	    cells, cell_pitch, pixels, pix_pitch);
}


__device__ Cell get_avg_cell(const Cell * cells, IVec2 coords, int pitch) {
	Cell res;

	for (int dy = -1; dy < 2; ++dy) {
		int y = coords.y + dy;
		for (int dx = -1; dx < 2; ++dx) {
			int x = coords.x + dx;
			res += *pitch_ptr(cells, {x, y}, pitch);
		}
	}

	return res / 9;
}

inline __device__ FVec2 carts(float theta) {
	FVec2 res;
	sincosf(theta, &res.y, &res.x);
	return res;
}

inline __device__ FVec2 moved(FVec2 src, float dir) {
	return src + (carts(dir) * params::sensor_distance);
}

inline __device__ Cell sample(Cell * cells, FVec2 coords, int pitch) {
	IVec2 i_coords = {static_cast<int>(coords.x), static_cast<int>(coords.y)};
	return *pitch_ptr(cells, i_coords, pitch);
}

__global__ void
agent_step(float * xs, float * ys, float * dirs, const Cell * cells, int pitch) {
	auto idx = idx_1();
	if (idx >= params::n_agents) return;

	auto coords = FVec2{xs[idx], ys[idx]};
	auto d = dirs[idx];

	// sense
	/*
	auto fl = sample(cells, moved(coords, d + params::sensor_angle_rad), pitch);
	auto c = sample(cells, moved(coords, d), pitch);
	auto fr = sample(cells, moved(coords, d - params::sensor_angle_rad), pitch);

	if (fl.x > c.x && fl.x > fr.x) {
	    d = fmodf(d + params::agent_turn_rad, 2 * M_PI);
	} else if (fr.x > fl.x && fr.x > c.x) {
	    d = fmodf(d - params::agent_turn_rad, 2 * M_PI);
	}
	*/

	// move
	coords += carts(d) * params::agent_move_distance;
	coords.x = fmodf(coords.x, params::width);
	coords.y = fmodf(coords.y, params::height);

	// Write back coordinates
	xs[idx] = coords.x;
	ys[idx] = coords.y;
	dirs[idx] = d;
}

__global__ void deposit(float * xs, float * ys, Cell * cells, int pitch) {
	auto idx = idx_1();
	if (idx >= params::n_agents) return;

	int x = static_cast<int>(xs[idx]);
	int y = static_cast<int>(ys[idx]);
	atomicAdd(&(pitch_ptr(cells, {x, y}, pitch)->x), params::deposit);
}

inline __device__ void decay(Cell * cell) { cell->x *= 0.9; }

__global__ void media_step(const Cell * prev, Cell * next, int pitch) {
	auto coords = idx_2();
	if (coords.x >= static_cast<int>(params::width)) return;
	if (coords.y >= static_cast<int>(params::height)) return;

	Cell * output = pitch_ptr(next, coords, pitch);
	Cell   c = get_avg_cell(prev, coords, pitch);
	// Cell c = *pitch_ptr(prev, coords, pitch);

	decay(&c);
	*output = c;
}

__host__ void step(
    Cell ** prev,
    Cell ** next,
    int     cell_pitch,
    float * x,
    float * y,
    float * dir,
    Pixel * pixels,
    int     pix_pitch) {
	agent_step<<<params::agent_grid_dim, params::agent_block_dim>>>(
	    x, y, dir, *prev, cell_pitch);

	deposit<<<params::agent_grid_dim, params::agent_block_dim>>>(
	    x, y, *prev, cell_pitch);

	media_step<<<params::cells_grid_dim, params::cells_block_dim>>>(
	    *prev, *next, cell_pitch);

	render(*next, cell_pitch, pixels, pix_pitch);

	Cell * tmp = *prev;
	*prev = *next;
	*next = tmp;
}
